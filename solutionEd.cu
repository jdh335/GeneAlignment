#include "hip/hip_runtime.h"
#include "htk.h"
#include <omp.h>
#include <iostream>
#include <fstream>
#include <map>
#include <tuple>
#include <string>
#include <limits>

using namespace std;

#define MATCH -3
#define INDEL 5
#define SUB 1

struct location {
    int i;
    int j;
};

struct align_data {
    float score;
    location back_pointer;
};

__global__ void align_kernel(char* seq1, char* seq2, align_data* align_matrix, int rows, int cols, int d, float INF) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = d - i;
    
    if (i < rows && j < cols && j >= 0) {
        float left = (i == 0) ? INF : align_matrix[(i - 1) * cols + j].score + INDEL;
        float top = (j == 0) ? INF : align_matrix[i * cols + (j - 1)].score + INDEL;
        float diagonal = (i == 0 || j == 0) ? INF : ((seq1[i] == seq2[j]) ? MATCH : SUB) + align_matrix[(i - 1) * cols + (j - 1)].score;

        align_data best;
        best.score = INF;

        if (left < best.score) best = {left, {i - 1, j}};
        if (top < best.score) best = {top, {i, j - 1}};
        if (diagonal < best.score) best = {diagonal, {i - 1, j - 1}};
        if (best.score == INF) best.score = 0;

        align_matrix[i * cols + j] = best;
    }
}

float align(string s1, string s2, bool banded, int align_length) {
    const float INF = numeric_limits<float>::infinity();

    string seq1 = "-" + s1.substr(0, align_length);
    string seq2 = "-" + s2.substr(0, align_length);

    int rows = seq2.length();
    int cols = seq1.length();

    align_data* align_matrix;
    hipMallocManaged(&align_matrix, rows * cols * sizeof(align_data));

    char* d_seq1;
    char* d_seq2;

    //managed memory added
    hipMallocManaged(&d_seq1, seq1.length() * sizeof(char));
    hipMallocManaged(&d_seq2, seq2.length() * sizeof(char));

    hipStream_t stream;
    hipStreamCreate(&stream);
    // asyn added
    hipMemcpyAsync(d_seq1, seq1.c_str(), seq1.length() * sizeof(char), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_seq2, seq2.c_str(), seq2.length() * sizeof(char), hipMemcpyHostToDevice, stream);

    dim3 threadsPerBlock(16);
    for (int d = 0; d < rows + cols - 1; ++d) {
        int numElements = min(d + 1, min(rows, cols));
        dim3 numBlocks((numElements + threadsPerBlock.x - 1) / threadsPerBlock.x);
        align_kernel<<<numBlocks, threadsPerBlock, 0, stream>>>(d_seq1, d_seq2, align_matrix, rows, cols, d, INF);
    }

    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);

    align_data result = align_matrix[(rows - 1) * cols + (cols - 1)];

    hipFree(d_seq1);
    hipFree(d_seq2);
    hipFree(align_matrix);

    return result.score;
}

int main(int argc, char* argv[]) {
    string line;
    string seq[8];
    string filename[8] = {"BCoV-ENT", "BCoV-LUN", "BCoV-Mebus", "BCoV-Quebec",
                          "Mouse_Hepatitis", "Murine_Hepatitis1", "Murine_Hepatitis2", "Murine_Hepatitis3"};

    for (int i = 0; i < 8; i++) {
        ifstream file("./sequences/" + filename[i] + ".txt");
        while (getline(file, line)) seq[i] += line;
    }

    htkTime_start(Compute, "Doing the computation");

    for (int i = 0; i < 8; i++) {
        for (int j = 0; j < 8; j++) {
            if (j >= i) {
                auto result = align(seq[i], seq[j], false, 30000);
                printf("Result: %f\n", result);
            }
        }
    }

    htkTime_stop(Compute, "Doing the computation");

    return 0;
}
